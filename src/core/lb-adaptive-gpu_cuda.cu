#include "hip/hip_runtime.h"
#include "config.hpp"

#ifdef LB_ADAPTIVE_GPU

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "lb-adaptive-gpu.hpp"
#include "lb-d3q19.hpp"
#include "utils.hpp"

LB_Parameters *d_lbpar = NULL;
LB_Model *d_lbmodel = NULL;
LB_Boundary *d_lb_boundaries;
lb_float *d_d3q19_lattice = NULL;
lb_float *d_d3q19_w = NULL;

void print_device_info() {
  int device = -1;
  hipGetDevice(&device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  printf("Device Number: %d\n", device);
  printf("  Device name: %s\n", prop.name);
  printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
  printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  printf("  Warp size: %i\n", prop.warpSize);
  printf("  Max memory pitch allowed: %i\n", prop.memPitch);
  printf("  Constant memory available: %i\n", prop.totalConstMem);
  printf("  Peak Memory Bandwidth (GB/s): %f\n",
         2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
  printf("  Number of Streaming Multiprocessors: %i\n",
         prop.multiProcessorCount);
  printf("  Maximum number of Threads per streaming multiprocessor: %i\n",
         prop.maxThreadsPerMultiProcessor);
  printf("  Maximum number of Threads per block: %i\n",
         prop.maxThreadsPerBlock);
  printf("  Maximum thread size: [%i, %i, %i]\n", prop.maxThreadsDim[0],
         prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
  printf("  Maximum grid size: [%i, %i, %i]\n", prop.maxGridSize[0],
         prop.maxGridSize[1], prop.maxGridSize[2]);
  printf("  Maximum amount of shared memory per block: %zu\n",
         prop.sharedMemPerBlock);
  printf("  Maximum amount of shared memory per streaming multiprocessor %zu\n",
         prop.sharedMemPerMultiprocessor);
  printf("\n");
}

void lbadapt_gpu_init() {
  print_device_info();

  if (d_d3q19_lattice == NULL) {
    CUDA_CALL(hipMalloc(&d_d3q19_lattice, sizeof(lb_float) * 3 * 19));
    CUDA_CALL(hipMemcpy(d_d3q19_lattice, d3q19_lattice,
                         sizeof(lb_float) * 19 * 3, hipMemcpyHostToDevice));
  }

  if (d_d3q19_w == NULL) {
    CUDA_CALL(hipMalloc(&d_d3q19_w, sizeof(lb_float) * 19));
    CUDA_CALL(hipMemcpy(d_d3q19_w, &d3q19_w, sizeof(lb_float) * 19,
                         hipMemcpyHostToDevice));
  }

  if (d_lbmodel == NULL) {
    CUDA_CALL(hipMalloc(&d_lbmodel, sizeof(LB_Model)));
    CUDA_CALL(hipMemcpy(d_lbmodel, &lbmodel, sizeof(LB_Model),
                         hipMemcpyHostToDevice));
  }

  if (d_lbpar == NULL) {
    CUDA_CALL(hipMalloc(&d_lbpar, sizeof(LB_Parameters)));
  }
  lbpar.agrid = (lb_float)P8EST_QUADRANT_LEN(lbpar.max_refinement_level) /
                ((lb_float)LBADAPT_PATCHSIZE * (lb_float)P8EST_ROOT_LEN);
  CUDA_CALL(hipMemcpy(d_lbpar, &lbpar, sizeof(LB_Parameters),
                       hipMemcpyHostToDevice));
}

void lbadapt_gpu_allocate_device_memory() {
  CUDA_CALL(
      hipMalloc(&d_lb_boundaries, n_lb_boundaries * sizeof(LB_Boundary)));
  CUDA_CALL(hipMemcpy(d_lb_boundaries, lb_boundaries,
                       n_lb_boundaries * sizeof(LB_Boundary),
                       hipMemcpyHostToDevice));

  assert(dev_local_real_quadrants == NULL);
  assert(dev_local_virt_quadrants == NULL);
  dev_local_real_quadrants = (lbadapt_payload_t **)malloc(
      sizeof(lbadapt_payload_t *) * P8EST_MAXLEVEL);
  dev_local_virt_quadrants = (lbadapt_payload_t **)malloc(
      sizeof(lbadapt_payload_t *) * P8EST_MAXLEVEL);
  for (int l = 0; l < P8EST_MAXLEVEL; ++l) {
    CUDA_CALL(hipMalloc(&dev_local_real_quadrants[l],
                         local_num_real_quadrants_level[l] *
                             sizeof(lbadapt_payload_t)));
    CUDA_CALL(hipMalloc(&dev_local_virt_quadrants[l],
                         local_num_virt_quadrants_level[l] *
                             sizeof(lbadapt_payload_t)));
  }
}

void lbadapt_gpu_deallocate_device_memory() {
  CUDA_CALL(hipFree(d_lb_boundaries));
  CUDA_CALL(hipFree(d_lbmodel));
  CUDA_CALL(hipFree(d_lbpar));
  CUDA_CALL(hipFree(d_d3q19_w));
  CUDA_CALL(hipFree(d_d3q19_lattice));

  if (dev_local_real_quadrants == NULL) {
    return;
  }
  for (int l = 0; l < P8EST_MAXLEVEL; ++l) {
    CUDA_CALL(hipFree(dev_local_real_quadrants[l]));
    CUDA_CALL(hipFree(dev_local_virt_quadrants[l]));
  }
  free(dev_local_real_quadrants);
  free(dev_local_virt_quadrants);
  dev_local_real_quadrants = NULL;
  dev_local_virt_quadrants = NULL;
}

// TODO: Use asynchronous memcpy
void lbadapt_gpu_copy_data_to_device(lbadapt_payload_t *source_real,
                                     lbadapt_payload_t *source_virt,
                                     int level) {
  if (source_real) {
    CUDA_CALL(hipMemcpy(dev_local_real_quadrants[level], source_real,
                         sizeof(lbadapt_payload_t) *
                             local_num_real_quadrants_level[level],
                         hipMemcpyHostToDevice));
  }
  if (source_virt) {
    CUDA_CALL(hipMemcpy(dev_local_virt_quadrants[level], source_virt,
                         sizeof(lbadapt_payload_t) *
                             local_num_virt_quadrants_level[level],
                         hipMemcpyHostToDevice));
  }
}

// TODO: Use asynchronous memcpy
void lbadapt_gpu_copy_data_from_device(lbadapt_payload_t *dest_real,
                                       lbadapt_payload_t *dest_virt,
                                       int level) {
  if (dest_real) {
    CUDA_CALL(hipMemcpy(dest_real, dev_local_real_quadrants[level],
                         sizeof(lbadapt_payload_t) *
                             local_num_real_quadrants_level[level],
                         hipMemcpyDeviceToHost));
  }
  if (dest_virt) {
    CUDA_CALL(hipMemcpy(dest_virt, dev_local_virt_quadrants[level],
                         sizeof(lbadapt_payload_t) *
                             local_num_virt_quadrants_level[level],
                         hipMemcpyDeviceToHost));
  }
}

__global__ void lbadapt_gpu_collide_calc_modes(lbadapt_payload_t *quad_data) {
  // clang-format off
  // mass mode
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 5] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18];

  // kinetic modes
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 6]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  // stress modes
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] =
     -quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 0] +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]) -
    2.0f * ((quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 5] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 6]) -
            (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) -
            (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]));

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  // kinetic modes
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] =
    -2.0f * (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] -
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] =
    -2.0f * (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] -
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] =
    -2.0f * (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 5] -
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 6]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] =
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 0] +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]) -
    2.0f * ((quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) +
            (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) +
            (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 5] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 6]));

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] =
    -(quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) +
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18] =
    -(quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 2]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 4]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16]) -
     (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18]) +
    2.0f * ((quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 5] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 6]) +
            (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 7] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 8]) +
            (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][ 9] +
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10]));
  // clang-format on
}

__global__ void lbadapt_gpu_collide_relax_modes(lbadapt_payload_t *quad_data,
                                                int level, lb_float h_max,
                                                LB_Parameters *d_lbpar) {
  lb_float rho, j[3], pi_eq[6];

  /** reconstruct real density */
  rho = quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
        (*d_lbpar).rho[0] * h_max * h_max * h_max;

  /** momentum density is redefined to include half-step of force action */
  j[0] = quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1];
  j[1] = quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2];
  j[2] = quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3];

  j[0] +=
      0.5 * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0];
  j[1] +=
      0.5 * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1];
  j[2] +=
      0.5 * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2];

  /** calculate equilibrium part of stress modes */
  pi_eq[0] = ((j[0] * j[0]) + (j[1] * j[1]) + (j[2] * j[2])) / rho;
  pi_eq[1] = ((j[0] * j[0]) - (j[1] * j[1])) / rho;
  pi_eq[2] =
      ((j[0] * j[0]) + (j[1] * j[1]) + (j[2] * j[2]) - 3.0f * (j[2] * j[2])) /
      rho;
  pi_eq[3] = (j[0] * j[1]) / rho;
  pi_eq[4] = (j[0] * j[2]) / rho;
  pi_eq[5] = (j[1] * j[2]) / rho;

  /** relax stress modes toward equilibrium */
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] =
      pi_eq[0] +
      d_lbpar->gamma_bulk[level] *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] -
           pi_eq[0]);
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] =
      pi_eq[1] +
      d_lbpar->gamma_shear[level] *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
           pi_eq[1]);
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] =
      pi_eq[2] +
      d_lbpar->gamma_shear[level] *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
           pi_eq[2]);
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] =
      pi_eq[3] +
      d_lbpar->gamma_shear[level] *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] -
           pi_eq[3]);
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] =
      pi_eq[4] +
      d_lbpar->gamma_shear[level] *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] -
           pi_eq[4]);
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] =
      pi_eq[5] +
      d_lbpar->gamma_shear[level] *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] -
           pi_eq[5]);

  /** relax ghost modes */
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] *=
      d_lbpar->gamma_odd[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] *=
      d_lbpar->gamma_odd[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] *=
      d_lbpar->gamma_odd[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] *=
      d_lbpar->gamma_odd[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] *=
      d_lbpar->gamma_odd[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] *=
      d_lbpar->gamma_odd[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] *=
      d_lbpar->gamma_even[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] *=
      d_lbpar->gamma_even[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18] *=
      d_lbpar->gamma_even[0];
}

// TODO: Implement
__global__ void
lbadapt_gpu_collide_thermalize_modes(lbadapt_payload_t *quad_data) {}

__global__ void lbadapt_gpu_collide_apply_forces(lbadapt_payload_t *quad_data,
                                                 int level, double h_max,
                                                 LB_Parameters *d_lbpar) {
  lb_float rho, u[3], C[6];

  /** reconstruct density */
  rho = quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
        d_lbpar->rho[0] * h_max * h_max * h_max;

  /** momentum density is redefined in case of external forces */
  u[0] =
      (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] * 0.5f *
       quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0]) /
      rho;
  u[1] =
      (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] * 0.5f *
       quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1]) /
      rho;
  u[2] =
      (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] * 0.5f *
       quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2]) /
      rho;

  C[0] = (1. + d_lbpar->gamma_bulk[level]) * u[0] *
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0] +
         1. / 3. * (d_lbpar->gamma_bulk[level] - d_lbpar->gamma_shear[level]) *
             (u[0] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[0] +
              u[1] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[1] +
              u[2] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[2]);
  C[2] = (1. + d_lbpar->gamma_bulk[level]) * u[1] *
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1] +
         1. / 3. * (d_lbpar->gamma_bulk[level] - d_lbpar->gamma_shear[level]) *
             (u[0] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[0] +
              u[1] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[1] +
              u[2] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[2]);
  C[5] = (1. + d_lbpar->gamma_bulk[level]) * u[2] *
             quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2] +
         1. / 3. * (d_lbpar->gamma_bulk[level] - d_lbpar->gamma_shear[level]) *
             (u[0] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[0] +
              u[1] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[1] +
              u[2] *
                  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z]
                      .force[2]);
  C[1] =
      0.5 * (1. + d_lbpar->gamma_shear[level]) *
      (u[0] * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1] +
       u[1] * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0]);
  C[3] =
      0.5 * (1. + d_lbpar->gamma_shear[level]) *
      (u[0] * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2] +
       u[2] * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0]);
  C[4] =
      0.5 * (1. + d_lbpar->gamma_shear[level]) *
      (u[1] * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2] +
       u[2] * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1]);

  /** update momentum modes */
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +=
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +=
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +=
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2];

  /** update stress modes */
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +=
      C[0] + C[2] + C[5];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] +=
      C[0] - C[2];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +=
      C[0] + C[2] - 2.0f * C[5];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] += C[1];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] += C[3];
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] += C[4];

/** reset external force */
#ifdef EXTERNAL_FORCES
  // unit conversion: force density
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0] =
      d_lbpar->prefactors[level] * d_lbpar->ext_force[0] * h_max * h_max *
      d_lbpar->tau * d_lbpar->tau;
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1] =
      d_lbpar->prefactors[level] * d_lbpar->ext_force[1] * h_max * h_max *
      d_lbpar->tau * d_lbpar->tau;
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2] =
      d_lbpar->prefactors[level] * d_lbpar->ext_force[2] * h_max * h_max *
      d_lbpar->tau * d_lbpar->tau;
#else  // EXTERNAL_FORCES
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[0] =
      (lb_float)0.0;
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[1] =
      (lb_float)0.0;
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].force[2] =
      (lb_float)0.0;
#endif // EXTERNAL_FORCES
}

__global__ void
lbadapt_gpu_collide_backtransform(lbadapt_payload_t *quad_data) {
  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][0] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][1] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18] -
      2.0f *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] +
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][2] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18] +
      2.0f *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] -
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][3] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18] -
      2.0f *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] +
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][4] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18] +
      2.0f *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] -
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][5] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] -
      2.0f *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] +
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] -
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][6] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] -
      2.0f *
          (quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] +
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] -
           quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18]);

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][7] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][8] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][9] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][10] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[7] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      2.0f * quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][11] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][12] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][13] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][14] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[1] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[8] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[10] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[13] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][15] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][16] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][17] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];

  quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].lbfluid[0][18] =
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[0] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[2] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[3] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[4] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[5] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[6] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[9] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[11] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[12] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[14] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[15] +
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[16] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[17] -
      quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].modes[18];
}

void lbadapt_gpu_execute_collision_kernel(int level) {

  dim3 blocks_per_grid(local_num_real_quadrants_level[level]);
  dim3 threads_per_block(LBADAPT_PATCHSIZE_HALO, LBADAPT_PATCHSIZE_HALO,
                         LBADAPT_PATCHSIZE_HALO);

  lb_float h_max = (lb_float)P8EST_QUADRANT_LEN(lbpar.max_refinement_level) /
                   ((lb_float)LBADAPT_PATCHSIZE * (lb_float)P8EST_ROOT_LEN);

  /** call kernels: calc modes, relax modes, thermalize modes, apply forces,
   *                backtransform */
  // TODO: smarter to put into a single kernel?
  lbadapt_gpu_collide_calc_modes<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level]);

  lbadapt_gpu_collide_relax_modes<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level], level, h_max, d_lbpar);

  lbadapt_gpu_collide_thermalize_modes<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level]); // stub only

  lbadapt_gpu_collide_apply_forces<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level], level, h_max, d_lbpar);

  lbadapt_gpu_collide_backtransform<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level]);
}

void lbadapt_gpu_execute_populate_virtuals_kernel(int level) {}

void lbadapt_gpu_execute_update_from_virtuals_kernel(int level) {}

__global__ void lbadapt_gpu_stream(lbadapt_payload_t *quad_data,
                                   LB_Model *d_lbmodel,
                                   lb_float *d_d3q19_lattice) {
  for (int i = 0; i < d_lbmodel->n_veloc; ++i) {
    // add 1 for halo offset
    quad_data
        ->patch[1 + threadIdx.x + (int)d_d3q19_lattice[3 * i + 0]]
               [1 + threadIdx.y + (int)d_d3q19_lattice[3 * i + 1]]
               [1 + threadIdx.z + (int)d_d3q19_lattice[3 * i + 2]]
        .lbfluid[1][i] =
        quad_data->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
            .lbfluid[0][i];
  }
}

void lbadapt_gpu_execute_streaming_kernel(int level) {
  dim3 blocks_per_grid(local_num_real_quadrants_level[level]);
  dim3 threads_per_block(LBADAPT_PATCHSIZE, LBADAPT_PATCHSIZE,
                         LBADAPT_PATCHSIZE);

  lbadapt_gpu_stream<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level], d_lbmodel, d_d3q19_lattice);

#if 0
  blocks_per_grid.x = local_num_virt_quadrants_level[level];

  lbadapt_gpu_stream<<<blocks_per_grid, threads_per_block>>>(
      dev_local_virt_quadrants[level], d_lbmodel, d_d3q19_lattice);
#endif // 0
}

__global__ void
lbadapt_gpu_bounce_back(lbadapt_payload_t *quad_data, lb_float h_max,
                        LB_Boundary *d_lb_boundaries, LB_Parameters *d_lbpar,
                        LB_Model *d_lbmodel, lb_float *d_d3q19_lattice,
                        lb_float *d_d3q19_w) {
  lb_float population_shift;
  /** if current quadrant is boundary: reset resting velocity to 0 and stream
   * all obtained velocities back to neighboring quadrants */
  if (quad_data->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
          .boundary) {
    quad_data->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
        .lbfluid[1][0] = (lb_float)0.0;
    /** bounce back to inverse velocity of current cell if neighboring cell is
     * boundary.
     * In the streaming phase this cell obtained populations that have to be
     * mirrored back into the original cell, i.e. lbfluid[1][i] has to be
     * written to lbfluid[1][inv(i)] of neighbor in direction inv(i).
     */
    for (int i = 1; i < d_lbmodel->n_veloc; i += 2) {
      // 2 step loop to avoid if statement
      // first step: inverse velocity is i + 1

      // calculate population shift from inflow/outflow boundary conditions
      population_shift = (lb_float)0.0;
      for (int l = 0; l < 3; l++) {
        population_shift -=
            h_max * h_max * h_max * d_lbpar->rho[0] * 2 *
            d_d3q19_lattice[3 * i + l] * d_d3q19_w[i] *
            d_lb_boundaries
                [quad_data
                     ->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
                     .boundary -
                 1].velocity[l] /
            d_lbmodel->c_sound_sq;
      }
      // sum up the force that is applied by the fluid
      for (int l = 0; l < 3; ++l) {
        d_lb_boundaries
            [quad_data->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
                 .boundary -
             1].force[l] +=
            (2 *
                 quad_data
                     ->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
                     .lbfluid[1][i] +
             population_shift) *
            d_d3q19_lattice[3 * i + l];
      }
      // do the actual bounce back step
      quad_data
          ->patch[1 + threadIdx.x + (int)d_d3q19_lattice[3 * (i + 1) + 0]]
                 [1 + threadIdx.y + (int)d_d3q19_lattice[3 * (i + 1) + 1]]
                 [1 + threadIdx.z + (int)d_d3q19_lattice[3 * (i + 1) + 2]]
          .lbfluid[1][i + 1] =
          quad_data->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
              .lbfluid[1][i] +
          population_shift;

      // second step: inverse velocity is i - 1
      // calculate population shift from inflow/outflow boundary conditions
      population_shift = (lb_float)0.0;
      for (int l = 0; l < 3; l++) {
        population_shift -=
            h_max * h_max * h_max * d_lbpar->rho[0] * 2 *
            d_d3q19_lattice[3 * i + l] * d_d3q19_w[i] *
            d_lb_boundaries
                [quad_data
                     ->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
                     .boundary -
                 1].velocity[l] /
            d_lbmodel->c_sound_sq;
      }
      // sum up the force that is applied by the fluid
      for (int l = 0; l < 3; ++l) {
        d_lb_boundaries
            [quad_data->patch[threadIdx.x][threadIdx.y][threadIdx.z].boundary -
             1].force[l] +=
            (2 *
                 quad_data
                     ->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
                     .lbfluid[1][i] +
             population_shift) *
            d_d3q19_lattice[3 * i + l];
      }
      // do the actual bounce back step
      quad_data
          ->patch[1 + threadIdx.x + (int)d_d3q19_lattice[3 * (i - 1) + 0]]
                 [1 + threadIdx.y + (int)d_d3q19_lattice[3 * (i - 1) + 1]]
                 [1 + threadIdx.z + (int)d_d3q19_lattice[3 * (i - 1) + 2]]
          .lbfluid[1][i - 1] =
          quad_data->patch[1 + threadIdx.x][1 + threadIdx.y][1 + threadIdx.z]
              .lbfluid[1][i] +
          population_shift;
    }
  }
}

void lbadapt_gpu_execute_bounce_back_kernel(int level) {
  dim3 blocks_per_grid(local_num_real_quadrants_level[level]);
  dim3 threads_per_block(LBADAPT_PATCHSIZE, LBADAPT_PATCHSIZE,
                         LBADAPT_PATCHSIZE);

  lb_float h_max = (lb_float)P8EST_QUADRANT_LEN(lbpar.max_refinement_level) /
                   ((lb_float)LBADAPT_PATCHSIZE * (lb_float)P8EST_ROOT_LEN);

  lbadapt_gpu_bounce_back<<<blocks_per_grid, threads_per_block>>>(
      dev_local_real_quadrants[level], h_max, d_lb_boundaries, d_lbpar,
      d_lbmodel, d_d3q19_lattice, d_d3q19_w);

#if 0
  blocks_per_grid.x = local_num_virt_quadrants_level[level];
  lbadapt_gpu_bounce_back<<<blocks_per_grid, threads_per_block>>>(
      dev_local_virt_quadrants[level], h_max, d_lb_boundaries, d_lbpar,
      d_lbmodel, d_d3q19_lattice, d_d3q19_w);
#endif // 0
}

// NOT LB-specific; visualize utilization of thread and block ids in vtk format
__global__ void visualize_threads_blocks(thread_block_container_t *a) {
  a[blockIdx.x].thread_idx[threadIdx.x][threadIdx.y][threadIdx.z] =
      LBADAPT_PATCHSIZE_HALO * LBADAPT_PATCHSIZE_HALO * threadIdx.z +
      LBADAPT_PATCHSIZE_HALO * threadIdx.y + threadIdx.x;
  a[blockIdx.x].block_idx[threadIdx.x][threadIdx.y][threadIdx.z] =
      LBADAPT_PATCHSIZE_HALO * LBADAPT_PATCHSIZE_HALO * blockIdx.z +
      LBADAPT_PATCHSIZE_HALO * blockIdx.y + blockIdx.x;
}

void show_blocks_threads(thread_block_container_t *data_host) {
  thread_block_container_t *data_dev;
  size_t data_size = sizeof(thread_block_container_t) * local_num_quadrants;

  CUDA_CALL(hipMalloc(&data_dev, data_size));

  dim3 blocks_per_grid(local_num_quadrants);
  dim3 threads_per_block(LBADAPT_PATCHSIZE_HALO, LBADAPT_PATCHSIZE_HALO,
                         LBADAPT_PATCHSIZE_HALO);

  visualize_threads_blocks<<<blocks_per_grid, threads_per_block>>>(data_dev);

  CUDA_CALL(hipMemcpy(data_host, data_dev, data_size, hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(data_dev));
}
#endif // LB_ADAPTIVE_GPU

#include "hip/hip_runtime.h"
#include "config.hpp"

#ifdef LB_ADAPTIVE_GPU
#include <hip/hip_runtime.h>

#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "lb-adaptive-gpu.hpp"

__global__ void simple_kernel(test_grid_t *a) {
  a->thread_idx[threadIdx.x][threadIdx.y][threadIdx.z] =
      (lb_float)LBADAPT_PATCHSIZE_HALO * (lb_float)LBADAPT_PATCHSIZE_HALO *
          (lb_float)threadIdx.z +
      (lb_float)LBADAPT_PATCHSIZE_HALO * (lb_float)threadIdx.y +
      (lb_float)threadIdx.x;
  a->block_idx[threadIdx.x][threadIdx.y][threadIdx.z] =
      (lb_float)LBADAPT_PATCHSIZE_HALO * (lb_float)LBADAPT_PATCHSIZE_HALO *
          (lb_float)blockIdx.z +
      (lb_float)LBADAPT_PATCHSIZE_HALO * (lb_float)blockIdx.y +
      (lb_float)blockIdx.x;
}

void test(test_grid_t *data_host) {
  test_grid_t *data_dev;
  size_t data_size = sizeof(test_grid_t) * local_num_quadrants;

  hipMalloc(&data_dev, data_size);

  dim3 blocks_per_grid(local_num_quadrants);
  dim3 threads_per_block(LBADAPT_PATCHSIZE_HALO, LBADAPT_PATCHSIZE_HALO,
                         LBADAPT_PATCHSIZE_HALO);

  simple_kernel<<<blocks_per_grid, threads_per_block>>>(data_dev);

  hipMemcpy(data_host, data_dev, data_size, hipMemcpyDeviceToHost);

  hipFree(data_dev);
}
#endif // LB_ADAPTIVE_GPU
